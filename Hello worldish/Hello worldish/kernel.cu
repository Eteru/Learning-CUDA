
#include "hip/hip_runtime.h"

#include <iostream>

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		c[i] = a[i] + b[i];
	}
}

int main(void)
{
	static const uint16_t SIZE = 1024;

	int a[SIZE], b[SIZE], c[SIZE];
	int *d_a, *d_b, *d_c;

	hipError_t err_a = hipMalloc(&d_a, SIZE * sizeof(int));
	hipError_t err_b = hipMalloc(&d_b, SIZE * sizeof(int));
	hipError_t err_c = hipMalloc(&d_c, SIZE * sizeof(int));
	// Need to check errors, not now 'tho

	for (int i = 0; i < SIZE; ++i) {
		a[i] = i;
		b[i] = i + 1;
	}

	hipMemcpy(d_a, &a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	VectorAdd<<<1, SIZE>>>(d_a, d_b, d_c, SIZE);

	hipMemcpy(&c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; ++i)
		std::cout << c[i] << std::endl;

	hipFree(&d_a);
	hipFree(&d_b);
	hipFree(&d_c);


	return 1;
}